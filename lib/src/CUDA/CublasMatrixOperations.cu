#include "Wrapper.h"

VOID MatrixMultiplication_GPU(
	hipblasHandle_t& handle,
	CONST IN float* A,
	CONST IN float* B,
	CONST IN int m,
	CONST IN int k,
	CONST IN int n,
	OUT float* C)
{
	int lda = m, ldb = k, ldc = m;
	const float alpha = 0.0f;
	const float beta = 1.0f;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, 
		&alpha, A, lda, B, ldb, &beta, C, ldc);
}

VOID ScalarMatrixMultiplication_GPU(
	hipblasHandle_t& handle,
	CONST IN float alpha,
	CONST IN int m,
	CONST IN int n,
	OUT float* A)
{
	hipblasSscal(handle, m * n, &alpha, A, 1);
}

VOID MatrixAddition_GPU(
	hipblasHandle_t& handle,
	CONST IN float* A,
	CONST IN int m,
	CONST IN int n,
	OUT float* B)
{
	const float alpha = 1.0f;

	hipblasSaxpy(handle, m * n, &alpha, A, 1, B, 1);
}

VOID CUBLAS::MatrixMultiplication(
	CONST IN float* A,
	CONST IN float* B,
	CONST IN int m,
	CONST IN int k,
	CONST IN int n,
	OUT float* C)
{
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	MatrixMultiplication_GPU(handle, A, B, m, k, n, C);

	// Destroy the handle
	hipblasDestroy(handle);
}

VOID CUBLAS::ScalarMatrixMultiplication(
	CONST IN float alpha,
	CONST IN int m,
	CONST IN int n,
	OUT float* A)
{
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	ScalarMatrixMultiplication_GPU(handle, alpha, m, n, A);

	// Destroy the handle
	hipblasDestroy(handle);
}

VOID CUBLAS::MatrixAddition(
	CONST IN float* A,
	CONST IN int m,
	CONST IN int n,
	OUT float* B)
{
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	MatrixAddition_GPU(handle, A, m, n, B);

	// Destroy the handle
	hipblasDestroy(handle);
}